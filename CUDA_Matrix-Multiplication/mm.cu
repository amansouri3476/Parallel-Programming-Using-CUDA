#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE

#include "mm.h"
	
#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may change the following values
#define TILEX 512
#define TILEY 2

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m,const int n) {
	// write your GPU kernel function here
float Pvalue = 0;
for (int k ; k<n; k++)
{
float Mdelement = ad[ty*n+k];
float Ndelement = bd[k*n + tx];
Pvalue += Mdelement*Ndelement;
}
cd[ty*n+tx] = Pvalue;
	// note that maximum # of threads per block is 1024
}
