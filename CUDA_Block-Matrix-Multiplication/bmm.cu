#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define TILE 16

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILE,n/TILE);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILE,TILE);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	// write your GPU kernel function here

}
